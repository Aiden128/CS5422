#include <hip/hip_runtime.h>


#include <memory>
#include <fstream>
#include <string>

const int BLOCK_SIZE = 16;
const int INF = 1073741823;

/* Default structure for graph */
struct graphAPSPTopology {
    unsigned int nvertex; // number of vertex in graph
    std::unique_ptr<int[]> graph; // graph matrix

    /* Constructor for init fields */
    graphAPSPTopology(int nvertex): nvertex(nvertex) {
        int size = nvertex * nvertex;
        graph = std::unique_ptr<int[]>(new int[size]());
    }
};

/**
 * CUDA handle error, if error occurs print message and exit program
*
* @param error: CUDA error status
*/
#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} \

/**
 * Blocked CUDA kernel implementation algorithm Floyd Wharshall for APSP
 * Dependent phase 1
 *
 * @param blockId: Index of block
 * @param nvertex: Number of all vertex in graph
 * @param pitch: Length of row in memory
 * @param graph: Array of graph with distance between vertex on device
 */
static __global__
void _blocked_fw_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) {
    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = BLOCK_SIZE * blockId + idy;
    const int v2 = BLOCK_SIZE * blockId + idx;

    int newPath;

    const int cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) {
        cacheGraph[idy][idx] = graph[cellId];
    } else {
        cacheGraph[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in block
    __syncthreads();

    #pragma unroll
    for (int u = 0; u < BLOCK_SIZE; ++u) {
        newPath = cacheGraph[idy][u] + cacheGraph[u][idx];

        // Synchronize before calculate new value
        __syncthreads();
        if (newPath < cacheGraph[idy][idx]) {
            cacheGraph[idy][idx] = newPath;
        }

        // Synchronize to make sure that all value are current
        __syncthreads();
    }

    if (v1 < nvertex && v2 < nvertex) {
        graph[cellId] = cacheGraph[idy][idx];
    }
}

/**
 * Blocked CUDA kernel implementation algorithm Floyd Wharshall for APSP
 * Partial dependent phase 2
 *
 * @param blockId: Index of block
 * @param nvertex: Number of all vertex in graph
 * @param pitch: Length of row in memory
 * @param graph: Array of graph with distance between vertex on device
 */
static __global__
void _blocked_fw_partial_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) {
    if (blockIdx.x == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    int v1 = BLOCK_SIZE * blockId + idy;
    int v2 = BLOCK_SIZE * blockId + idx;

    __shared__ int cacheGraphBase[BLOCK_SIZE][BLOCK_SIZE];

    // Load base block for graph and predecessors
    int cellId = v1 * pitch + v2;

    if (v1 < nvertex && v2 < nvertex) {
        cacheGraphBase[idy][idx] = graph[cellId];
    } else {
        cacheGraphBase[idy][idx] = INF;
    }

    // Load i-aligned singly dependent blocks
    if (blockIdx.y == 0) {
        v2 = BLOCK_SIZE * blockIdx.x + idx;
    } else {
   // Load j-aligned singly dependent blocks
        v1 = BLOCK_SIZE * blockIdx.x + idy;
    }

    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    // Load current block for graph and predecessors
    int currentPath;

    cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) {
        currentPath = graph[cellId];
    } else {
        currentPath = INF;
    }
    cacheGraph[idy][idx] = currentPath;

    // Synchronize to make sure the all value are saved in cache
    __syncthreads();

    int newPath;
    // Compute i-aligned singly dependent blocks
    if (blockIdx.y == 0) {
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraphBase[idy][u] + cacheGraph[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }
            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    } else {
    // Compute j-aligned singly dependent blocks
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraph[idy][u] + cacheGraphBase[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }

            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    }

    if (v1 < nvertex && v2 < nvertex) {
        graph[cellId] = currentPath;
    }
}

/**
 * Blocked CUDA kernel implementation algorithm Floyd Wharshall for APSP
 * Independent phase 3
 *
 * @param blockId: Index of block
 * @param nvertex: Number of all vertex in graph
 * @param pitch: Length of row in memory
 * @param graph: Array of graph with distance between vertex on device
 */
static __global__
void _blocked_fw_independent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) {
    if (blockIdx.x == blockId || blockIdx.y == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = blockDim.y * blockIdx.y + idy;
    const int v2 = blockDim.x * blockIdx.x + idx;

    __shared__ int cacheGraphBaseRow[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int cacheGraphBaseCol[BLOCK_SIZE][BLOCK_SIZE];

    int v1Row = BLOCK_SIZE * blockId + idy;
    int v2Col = BLOCK_SIZE * blockId + idx;

    // Load data for block
    int cellId;
    if (v1Row < nvertex && v2 < nvertex) {
        cellId = v1Row * pitch + v2;

        cacheGraphBaseRow[idy][idx] = graph[cellId];
    }
    else {
        cacheGraphBaseRow[idy][idx] = INF;
    }

    if (v1  < nvertex && v2Col < nvertex) {
        cellId = v1 * pitch + v2Col;
        cacheGraphBaseCol[idy][idx] = graph[cellId];
    }
    else {
        cacheGraphBaseCol[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in virtual block
   __syncthreads();

   int currentPath;
   int newPath;

   // Compute data for block
   if (v1  < nvertex && v2 < nvertex) {
       cellId = v1 * pitch + v2;
       currentPath = graph[cellId];

        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
           newPath = cacheGraphBaseCol[idy][u] + cacheGraphBaseRow[u][idx];
           if (currentPath > newPath) {
               currentPath = newPath;
           }
       }
       graph[cellId] = currentPath;
   }
}

/**
 * Allocate memory on device and copy memory from host to device
 * @param dataHost: Reference to unique ptr to graph data with allocated fields on host
 * @param graphDevice: Pointer to array of graph with distance between vertex on device
 *
 * @return: Pitch for allocation
 */
static
size_t _cudaMoveMemoryToDevice(const std::unique_ptr<graphAPSPTopology>& dataHost, int **graphDevice) {
    size_t height = dataHost->nvertex;
    size_t width = height * sizeof(int);
    size_t pitch;

    // Allocate GPU buffers for matrix of shortest paths d(G) and predecessors p(G)
    HANDLE_ERROR(hipMallocPitch(graphDevice, &pitch, width, height));

    // Copy input from host memory to GPU buffers and
    HANDLE_ERROR(hipMemcpy2D(*graphDevice, pitch,
            dataHost->graph.get(), width, width, height, hipMemcpyHostToDevice));

    return pitch;
}

/**
 * Copy memory from device to host and free device memory
 *
 * @param graphDevice: Array of graph with distance between vertex on device
 * @param dataHost: Reference to unique ptr to graph data with allocated fields on host
 * @param pitch: Pitch for allocation
 */
static
void _cudaMoveMemoryToHost(int *graphDevice, const std::unique_ptr<graphAPSPTopology>& dataHost, size_t pitch) {
    size_t height = dataHost->nvertex;
    size_t width = height * sizeof(int);

    HANDLE_ERROR(hipMemcpy2D(dataHost->graph.get(), width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(graphDevice));
}

/**
 * Blocked implementation of Floyd Warshall algorithm in CUDA
 *
 * @param data: unique ptr to graph data with allocated fields on host
 */
void cudaBlockedFW(const std::unique_ptr<graphAPSPTopology>& dataHost) {
    HANDLE_ERROR(hipSetDevice(0));
    int nvertex = dataHost->nvertex;
    int *graphDevice;
    size_t pitch = _cudaMoveMemoryToDevice(dataHost, &graphDevice);

    dim3 gridPhase1(1 ,1, 1);
    dim3 gridPhase2((nvertex - 1) / BLOCK_SIZE + 1, 2 , 1);
    dim3 gridPhase3((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1 , 1);
    dim3 dimBlockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

    int numBlock = (nvertex - 1) / BLOCK_SIZE + 1;

    for(int blockID = 0; blockID < numBlock; ++blockID) {
        // Start dependent phase
        _blocked_fw_dependent_ph<<<gridPhase1, dimBlockSize>>>
                (blockID, pitch / sizeof(int), nvertex, graphDevice);

        // Start partially dependent phase
        _blocked_fw_partial_dependent_ph<<<gridPhase2, dimBlockSize>>>
                (blockID, pitch / sizeof(int), nvertex, graphDevice);

        // Start independent phase
        _blocked_fw_independent_ph<<<gridPhase3, dimBlockSize>>>
                (blockID, pitch / sizeof(int), nvertex, graphDevice);
    }

    // Check for any errors launching the kernel
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    _cudaMoveMemoryToHost(graphDevice, dataHost, pitch);
}

// void Write_file(const char *filename, int *AdjMatrix, const int &num_vertex) {
//     std::ofstream out_file(filename);
//     for (int i = 0; i < num_vertex; i++) {
//         out_file.write((char *)&AdjMatrix[i * num_vertex],
//                        sizeof(int) * num_vertex);
//     }
//     out_file.close();
// }

void Write_file(const std::string &filename, const std::unique_ptr<graphAPSPTopology> &data) {
    std::ofstream out_file(filename);
    for (int i = 0; i < data->nvertex; ++i) {
        out_file.write((char *)&data->graph[i * data->nvertex], sizeof(int) * data->nvertex);
    }
    out_file.close();
}

int main (int argc, char **argv) {
    std::fstream file;
    int num_vertex = 0;
    int num_edge = 0;
    int src = 0;
    int dest = 0;
    int weight = 0;
    std::string in_filename = argv[1];

    file.open(in_filename, std::ios::in | std::ios::binary);
    file.read((char *)&num_vertex, sizeof(num_vertex));
    file.read((char *)&num_edge, sizeof(num_edge));
    std::unique_ptr<graphAPSPTopology> AdjMatrix;
    AdjMatrix = std::unique_ptr<graphAPSPTopology>(new graphAPSPTopology(num_vertex));

    std::fill_n(AdjMatrix->graph.get(), num_vertex * num_vertex, INF);  
    for (int i = 0; i < num_vertex; i++) {
        AdjMatrix->graph[i * num_vertex + i] = 0;
    }

    int *tmp(new int[num_edge * 3]);
    file.read((char *)tmp, sizeof(int) * num_edge * 3);
    for (int i = 0; i < num_edge; ++i) {
        src = tmp[i * 3];
        dest = tmp[i * 3 + 1];
        weight = tmp[i * 3 + 2];
        int idx(src * num_vertex + dest);
        AdjMatrix->graph[idx] = weight;
    }
    file.close();

    return 0;
}